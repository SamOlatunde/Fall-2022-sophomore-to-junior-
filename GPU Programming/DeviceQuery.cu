#include<stdio.h>
#include<hip/hip_runtime.h>


int main()
{
    
   int dev_count;
   hipGetDeviceCount(&dev_count);
    
   hipDeviceProp_t dev_prop;
   for (int i =0; i < dev_count; i++)
   {
     hipGetDeviceProperties(&dev_prop, i);
     printf("%d:  %d\n", i, dev_prop.sharedMemPerBlock);
   }
   
   printf("%u", sizeof(float));



    return 0;
}