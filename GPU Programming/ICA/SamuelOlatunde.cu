// Samuel Olatunde 
// GPU Programming 
// ICA 1
#include<stdio.h>
#include<hip/hip_runtime.h>

// prototypes 
__global__
void arrPoduct1(int *d_A, int * d_B, int * d_C, int n);

__global__
void arrProduct2(int *d_A, int * d_B, int * d_C, int n);

int main()
{
    // variable and constant declarations 
    const int N = 10240;

    int A[N], B[N], C[N];
    int *d_A, * d_B, * d_C;
    int size = N * sizeof(int);

    // loop to initialize A with approprate values 
    for (int i = 0; i < N; i++)
    {
        A[i] = 2 * i;
    }
      
      

    // loop to initialize B with approprate values 
    for (int i = 0; i < N; i++)
    {
        B[i] = (2 * i) +1;
    }
    
    
    hipMalloc((void**) &d_A, size);
    hipMemcpy(d_A, A, size,hipMemcpyHostToDevice);

    hipMalloc((void**) &d_B, size);
    hipMemcpy(d_B, B, size,hipMemcpyHostToDevice);

    hipMalloc((void**) &d_C, size);

    dim3 dimGrid( 2, 1, 1);
    dim3 dimBlock(1024 , 1, 1);
    
    // First Kernel Launch
    arrPoduct1<<<dimGrid, dimBlock>>>(d_A,d_B,d_C, N);
    
    hipMemcpy(C, d_C, size,hipMemcpyDeviceToHost);

    printf("2  Blocks (first, last) = ("); 
    printf("%d ", C[0]);
    printf(", ");
    printf("%d", C[N-1]);
    printf(")\n");

    dim3 dimGrid1( 10, 1, 1);
    dim3 dimBlock1(1024 , 1, 1);

    // Second KErnel Launch
    arrProduct2<<<dimGrid1, dimBlock1>>> (d_A, d_B, d_C, N);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    
    printf("10 Blocks (first, last) = ("); 
    printf("%d ", C[0]);
    printf(", ");
    printf("%d", C[N-1]);
    printf(")\n");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    return 0;
}


// This function uses threads to compute the product of two 
// arrays and places the result in a third array 
__global__
void arrPoduct1(int * d_A, int * d_B, int * d_C, int n)
{
    // maps local index to global index
   int i = blockDim.x * blockIdx.x + threadIdx.x;

   // computes the product
   if (i < n) d_C[i] = d_A[i] * d_B[i];
}

// This function uses threads to compute the product of two 
// arrays and places the result in a third array 
__global__
void arrProduct2(int *d_A, int * d_B, int * d_C, int n)
{
   int i = blockDim.x * blockIdx.x + threadIdx.x;

   if (i < n) d_C[i] = d_A[i] * d_B[i];
}