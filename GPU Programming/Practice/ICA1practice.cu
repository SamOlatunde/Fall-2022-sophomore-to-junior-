#include "hip/hip_runtime.h"
#include <stdio.h>
#include<hip/hip_runtime.h>
using namespace std;

// prototypes 
__global__
void vecAddKernel(float * d_A, float * d_B, float * d_C, int n);
void vecAdd(float * h_A, float * h_B, float * h_C, int n);


int main()
{
    int size = 5;
    float A[size] = {2,2,2,2,2}, B[size] = {2,2,2,2,2}, C[size];

    vecAdd(A,B,C, size);

    for (int i = 0; i < size; i++)
    {
        printf("%f\t", C[i]);
    }    

    return 0;
}

// Kernel fucntion, n is the numbe o f elements in each array
__global__
void vecAddKernel(float * d_A, float * d_B, float * d_C, int n)
{
    // mapping local index to global index 
    int i = blockDim.x * blockIdx.x + threadIdx.x;
     
    if (i < n) 
      d_C[i] = d_A[i] + d_B[i];

}


void vecAdd(float * h_A, float * h_B, float * h_C, int n)
{
    float *d_A,* d_B, *d_C;

    // calculates number of bytees needed to store the array
    int size = n * sizeof(float);

    // allocates memory for d_A in the global memory and transfers data from main memory to global memory
    hipMalloc((void**) &d_A, size);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    // allocates memory for d_C in the global memory and transfers data from main memory to global memory
    hipMalloc((void**) &d_B, size);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // allocates memory for d_C in the global memory 
    hipMalloc((void**) & d_C, size);

    // specifies the grid dimensions
    dim3 dimGrid(1,1,1);

    // specifies block dimensions
    dim3 dimBlock(n,1,1);

    // kernel lauch statement 
    vecAddKernel<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, n);
    

    // copys the answer back to main memory 
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // frees up global memory 
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);



}