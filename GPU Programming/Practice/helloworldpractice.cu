#include<stdio.h>
#include<hip/hip_runtime.h>

const int N = 7;

__global__
void helloWorld(char * a, int * b);

int main()
{
    char a[N] = "Hello ";
    int b[N] = {15,10,6,0,-11,1,0};
    

    char * ad;
    int * bd;
    const int csize = N * sizeof(char);
    const int isize = N * sizeof(int);
    
    printf("%s", a);

    hipMalloc((void**) &ad, csize);
    hipMalloc((void**) &bd, isize);

    hipMemcpy(ad, a, csize, hipMemcpyHostToDevice);
    hipMemcpy(bd,b, isize, hipMemcpyHostToDevice);

    dim3 dimGrid(1,1,1);
    dim3 dimBlock(N,1,1);

    helloWorld<<<dimGrid, dimBlock >>>(ad,bd);

    hipMemcpy(a,ad,csize,hipMemcpyDeviceToHost);
    hipFree(ad);
    hipFree(bd);

    printf("%s", a);

    return 0;
}

__global__
void helloWorld(char * a, int * b)
{
   int i = blockDim.x * blockIdx.x +threadIdx.x;
   if(i < N) a[i]+=b[i];
}
