//*****************************************************************
// Assignment #2
// Name: Samuel Olatunde , and Sunil Rasaily 
// GPU Programming Date: Date of Submission (11/04/2022)
//******************************************************************
// This Program Implements a parallel CUDA GPU solution to the 
// computationally intensive Cooley-Tukey FFT Algorithm.The 
// parallel implmentation takes the complexity of the algorithm 
// an order of magnitude lower from O(n^2) to O(n). The program 
// uses a struct to store complex numbers. Input is hard coded and 
// output is sent to a file. 
//******************************************************************
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>
#define N 8192
#define tileWidth 1024

// struct to handle complex numbers
struct complexNum
{
  float real;
  float imag;
};

// Function Prototype
__global__ 
void calcCooleyTukey(struct complexNum * d_FFT, 
                                 struct complexNum * d_timeDomain);
 
int main()
{
    // host Variable declaration
    struct complexNum FFT[N];
    struct complexNum timeDomain[N];

    //Hard Coding first 8 entries of time domain 
    timeDomain[0].real = 3.6;
    timeDomain[0].imag = 2.6;
    timeDomain[1].real = 2.9;
    timeDomain[1].imag = 6.3;
    timeDomain[2].real = 5.6;
    timeDomain[2].imag = 4;
    timeDomain[3].real = 4.8;
    timeDomain[3].imag = 9.1;
    timeDomain[4].real = 3.3;
    timeDomain[4].imag = 0.4;
    timeDomain[5].real = 5.9;
    timeDomain[5].imag = 4.8;
    timeDomain[6].real = 5;
    timeDomain[6].imag = 2.6;
    timeDomain[7].real = 4.3;
    timeDomain[7].imag = 4.1;

    // initializing the other entries of time domain to 0
    for (int i = 8; i < N; i++)
    {
        timeDomain[i].real = 0;
        timeDomain[i].imag = 0;
    }
    
    // device variables 
    struct complexNum * d_FFT;
    struct complexNum * d_timeDomain;
    int size = N* sizeof(complexNum);
    
    // allocates global memory for device variables
    hipMalloc((void**) & d_FFT, size);
    hipMalloc((void**) & d_timeDomain, size);
    
    // Copies data from host to device 
    hipMemcpy(d_timeDomain, timeDomain, size, hipMemcpyHostToDevice);

    //Sets grid and block dimensions
    dim3 dimGrid(4,1,1);
    dim3 dimBlock(1024,1,1);

    //Kernel Lauch
    calcCooleyTukey<<<dimGrid, dimBlock>>>(d_FFT, d_timeDomain);

    // copies result from device to host 
    hipMemcpy(FFT, d_FFT, size, hipMemcpyDeviceToHost);
    
    //frees allocated memory
    hipFree(d_FFT);
    hipFree(d_timeDomain);
    
    // Prints the first 8 samples
    printf("TOTAL PROCESSED SAMPLES: %d\n", N);
    printf("===========================================\n");
    printf("XR[0]: %f\t", FFT[0].real);
    printf("XI[0]: %f\n", FFT[0].imag);
    printf("===========================================\n");
    for(int i =1; i < 8; i++)
    {
       printf("XR[%d]: %f\tXI[%d]: %f\n", i, FFT[i].real,
                                              i, FFT[i].imag);
    }
    printf("===========================================\n\n");
    
    for(int i =4096; i < 4096+8; i++)
    {
       printf("XR[%d]: %f\tXI[%d]: %f\n", i, FFT[i].real,
                                              i, FFT[i].imag);
    }
    printf("===========================================\n\n");
    
    return 0; 
}

//*******************************************************************
// Name::calcCooleyTukey()
// Parameters: 2 complexNum Pointers
// This kernel function computes the FFT for each value of k. The 
// kernel is written for one dimensional grid and blocks, and is 
// written with the assumption that it is launched with a grid size
// that is half of the size of the data set. Each thread takes 
// care of 1 FFT coeficent. Due to the frequency of access of 
// d_timeDomain and the fact that its values don't change throughout 
// execution, the kernel uses the idea of tiling, and shared memory, 
// to reduce the traffic in global memory. This way, each block only 
// has to perform the long latency operation of accessing each 
// d_timeDomain element once, which is when the threads bring the 
// values of d_timeDomain, to shared memory. In addition, the kernel
// uses barrier synchronization, to ensure that all threads have 
// finished a phase before moving on so as to avoid inconsistentcies 
// in our result. 
//********************************************************************
__global__ 
void calcCooleyTukey(struct complexNum * d_FFT, 
                                     struct complexNum * d_timeDomain)
{
    // local to global index mapping 
    int k = blockDim.x * blockIdx.x + threadIdx.x;
    int timeDomainElementsPerThread = N / blockDim.x;
   
   //Allocates shared memory 
   __shared__ struct complexNum tile[tileWidth];
    
    // Variables local to each thread
    float real = 0.0, imag = 0.0, theta = 0.0;
    
    // varibales to compute the other half of array
    float real2ndHalf = 0.0, imag2ndHalf = 0.0, theta2ndHalf = 0.0;

    // Shared memory version  
    // for a given processing window, copy the value of time
    // domain to tile, and compute the values of real and 
    // imag, then move processing window and accumulate 
    // values in real and imag until we have run through the 
    // whole array 
    for (int i = 0; i < timeDomainElementsPerThread; i++)
    {
        tile[threadIdx.x] = d_timeDomain[threadIdx.x + 
                                                      (i* blockDim.x)];
        __syncthreads(); 
        
        for( int n = 0; n < tileWidth; n++)
        {
            //Computation for upper half of d_FFT
            // computes theta for each element of timeDomain.
            // timeDomain index goes up to N, so we need to map
            // the tile index to timeDomain index when we are trying 
            // to compute theta
            theta = (2 * M_PI * (n+ (i* blockDim.x)) * k)/ N;

            //compute and accumulate result in real and imag 
            real += (tile[(n)].real * cos(theta)) +
                                (tile[(n)].imag * sin(theta));
            imag += (-(tile[(n)].real)* sin(theta)) + 
                                (tile[(n)].imag * cos(theta)); 
           
           //Computation for lower half of d_FFT
           theta2ndHalf = (2 * M_PI * (n+ (i* blockDim.x)) * 
                                                      (k+(N/2)))/ N;

           real2ndHalf += (tile[(n)].real * cos(theta2ndHalf)) +
                                (tile[(n)].imag * sin(theta2ndHalf));

            imag2ndHalf +=  (-(tile[(n)].real)* sin(theta2ndHalf)) + 
                                (tile[(n)].imag * cos(theta2ndHalf)); 
        }
        __syncthreads();
    }

    // store result at appropriate location
    d_FFT[k].real = real;
    d_FFT[k].imag = imag;

    d_FFT[k + (N/2)].real = real2ndHalf;
    d_FFT[k + (N/2)].imag = imag2ndHalf;

    /*Orginal Version, doesnt Implement shared memory
    for( int n = 0; n < (N/2); n++)
    {
        // computes theta for even-index part of  d_timeDomain
        theta = (2 * M_PI * (2 * n) * k)/ N;

        // compute the DFT of even-indexed part of d_timeDomain, 
        // and accumulate it in real and imag
        real += (d_timeDomain[(2* n)].real * cos(theta)) +
                               (d_timeDomain[(2 * n)].imag *
                                                          sin(theta));
        imag += (-(d_timeDomain[(2*n)].real)* sin(theta)) + 
                               (d_timeDomain[(2 * n)].imag * 
                                                           cos(theta)); 

        // computes theta for even-index part of  d_timeDomain
        theta = (2 * M_PI * ((2 * n) + 1) * k)/ N;

        // compute the DFT of odd-indexed part of d_timeDomain, 
        // and accumulate it in real and imag
        real += (d_timeDomain[(2* n) + 1].real * cos(theta)) + 
                               (d_timeDomain[(2 * n) + 1].imag * 
                                                           sin(theta));
        imag += (-(d_timeDomain[(2*n) + 1].real)* sin(theta)) +  
                               (d_timeDomain[(2 * n) + 1].imag *
                                                          cos(theta)); 
    
        theta2ndHalf = (2 * M_PI * (2 * n) * (k+(N/2)))/ N;

        real2ndHalf += (d_timeDomain[(2 * n)].real *
                           cos(theta2ndHalf)) +
                            (d_timeDomain[(2 * n)].imag *
                                            sin(theta2ndHalf));

        imag2ndHalf +=  (-(d_timeDomain[(2 * n)].real)*
                               sin(theta2ndHalf)) + 
                                    (d_timeDomain[(2 * n)].imag
                                          * cos(theta2ndHalf)); 

        theta2ndHalf = (2 * M_PI * (n+ ((2 * n) + 1) *
                                                   (k+(N/2))))/ N;

        real2ndHalf += (d_timeDomain[((2 * n) + 1)].real * 
                                 cos(theta2ndHalf)) +
                                    (d_timeDomain[((2 * n) + 1)].imag 
                                                   * sin(theta2ndHalf));

        imag2ndHalf +=  (-(d_timeDomain[((2 * n) + 1)].real)* 
                         sin(theta2ndHalf)) + 
                          (d_timeDomain[((2 * n) + 1)].imag *
                                                  cos(theta2ndHalf)); 
    }

    // store result at appropriate location
    d_FFT[k].real = real;
    d_FFT[k].imag = imag;

    d_FFT[k + (N/2)].real = real2ndHalf;
    d_FFT[k + (N/2)].imag = imag2ndHalf;*/
}
