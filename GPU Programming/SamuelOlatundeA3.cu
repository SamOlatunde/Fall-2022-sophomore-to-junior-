//*****************************************************************
//                   Assignment #3
//                    Names: Samuel Olatunde 
//                    GPU PRogramming
//                    Date: 10/05/2022
//*****************************************************************
//   This is a cuda c program that implements uses 
//   3 kernels calculate the product of two same size arrays, A
//   and B, then stores the result in a third array C
//*****************************************************************
#include<stdio.h>
#include<hip/hip_runtime.h>

// Kernel Prototypes 
__global__ 
void notCyclicArrProduct1(int * d_A, int * d_B, int * d_C, int N);
__global__
void cyclicArrProduct1(int * d_A, int * d_B, int * d_C, int N);
__global__
void arrProduct2(int * d_A, int * d_B, int * d_C);

int main()
{
    // Constant/variable declaration/initializations 
    const int N = 10240;
    int A[N],B[N],C[N];
    int * d_A, * d_B, * d_C;
    int size = N * sizeof(int);

    // Initializes array A with even sequence (2n)
    for (int i = 0; i < N; i++)
    {
        A[i] = 2 * i;
    }

    // Initializes array B with odd sequence (2n+1)
    for (int i = 0; i < N; i++)
    {
        B[i] = (2* i) + 1;
    }
    
    // allocates memory for A in the global memory, saves the
    // address in device variable d_A and transfers data from
    // main memory to global memory
    hipMalloc((void**) &d_A, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    
    // allocates memory for B in the global memory, saves the
    // address in device variable d_B and transfers data from
    // main memory to global memory
    hipMalloc((void **) &d_B, size);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // allocates memory for C in the global memory, saves the
    // address in device variable d_A 
    hipMalloc((void **) &d_C, size);
    
    // grid and block dimension specifications
    dim3 dimGrid(2,1,1);
    dim3 dimBlock(1024,1,1);

    // Kernel Launched with 2 blocks of 1024 threads each 
    notCyclicArrProduct1<<<dimGrid,dimBlock>>>(d_A,d_B,d_C, N);
    
    // copies data for C from global memory to main memory
    hipMemcpy(C,d_C, size, hipMemcpyDeviceToHost);
    
    // frees up global memory 
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // output statment 
    printf("2 Blocks - Not Cyclic ( C[0], C[10239] ) = (");
    printf("%d", C[0]);
    printf(", ");
    printf("%d", C[10239]);
    printf(")\n");
    
    // allocates memory for A in the global memory, saves the
    // address in device variable d_A and transfers data from
    // main memory to global memory
    hipMalloc((void**) & d_A, size);
    hipMemcpy(d_A,A,size, hipMemcpyHostToDevice);

    // allocates memory for B in the global memory, saves the
    // address in device variable d_B and transfers data from
    // main memory to global memory
    hipMalloc((void**) & d_B, size);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // allocates memory for C in the global memory, saves the
    // address in device variable d_C 
    hipMalloc((void**) & d_C, size);
    
    // Kernel launched with 2 blocks of 1024 threads each 
    cyclicArrProduct1<<< dimGrid, dimBlock>>>(d_A,d_B,d_C,N);

    // copies data for C from global memory to main memory
    hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);

    // frees up global memory 
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    printf("2 Blocks - Cyclic ( C[0], C[10239] ) = (");
    printf("%d", C[0]);
    printf(", ");
    printf("%d", C[10239]);
    printf(")\n");

    // allocates memory for A in the global memory, saves the
    // address in device variable d_A and transfers data from
    // main memory to global memory
    hipMalloc((void**) & d_A, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

    // allocates memory for B in the global memory, saves the
    // address in device variable d_B and transfers data from
    // main memory to global memory
    hipMalloc((void**) & d_B, size);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    
    // copies data for C from global memory to main memory
    hipMalloc((void**) & d_C, size);
    
    // grid and block specifications 
    dim3 dimGrid1(10,1,1);
    dim3 dimBlock1(1024,1,1);
    
    // Kernel Launched with 10 blocks of 1024 threads each
    arrProduct2<<< dimGrid1, dimBlock1 >>> (d_A, d_B, d_C);
    
    // copies data for C from global memory to main memory
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // frees up global memory 
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // output Statment 
    printf("10 Blocks -( C[0], C[10239] ) = (");
    printf("%d", C[0]);
    printf(", ");
    printf("%d", C[10239]);
    printf(")\n");

    return 0;
}


//********************************************************************
//                     Name:: notCyclicArrProduct1
//                    Parameters: 3 device pointers to arrays, 
//                                1 int for the size of each array 
//   This Kernel computes the product by noncyclic data partitioning 
//********************************************************************
__global__ 
void notCyclicArrProduct1(int * d_A, int * d_B, int * d_C, int N)
{
   int AmtOfDataPerThreadBlock = N / gridDim.x;
   int nonCyclicIndex;
   int AmtOfDataPerThread = AmtOfDataPerThreadBlock/ blockDim.x;

   for (int i = 0; i < AmtOfDataPerThread; i++)
   {
       nonCyclicIndex = (blockIdx.x * blockDim.x  * AmtOfDataPerThread 
                         + threadIdx.x) + (i * blockDim.x);

       d_C[nonCyclicIndex] = d_A[nonCyclicIndex] * d_B[nonCyclicIndex];
   }
}

//********************************************************************
//                     Name:: cyclicArrProduct1
//                    Parameters: 3 device pointers to arrays, 
//                                1 int for the size of each array 
//    This kernel computes the product by cyclic data partitioning, 
//    i.e. each thread block computes as many theadBlocks sized 
//    chunks of data as it needs to in order to process all data but 
//    in a cyclic manner. So if we had two thread blocks of size n 
///    and N data elements to process block 
//********************************************************************
__global__
void cyclicArrProduct1(int * d_A, int * d_B, int * d_C, int N)
{
   // local to global index mapping 
   int index = blockIdx.x * blockDim.x + threadIdx.x;

   // calculates the total number of threads in the grid 
   int totalThreads = gridDim.x * blockDim.x;
   int cyclicIndex; 

   // calculates the times we are reusing a thread
   int numberOfElementsPerThread = N/totalThreads;
 
   for (int i = 0; i < numberOfElementsPerThread; i++)
   {
     cyclicIndex = index + (i * totalThreads);

     d_C[cyclicIndex] = d_A[cyclicIndex] * d_B[cyclicIndex];
   }   
}

//********************************************************************
//                     Name:: notCyclicArrProduct1
//                    Parameters: 3 device pointers to arrays, 
//    This kernel is written with the assumption that it is launched 
//    with sufficent blocks and  threads the porcess all available
//    data                            
//********************************************************************
__global__
void arrProduct2(int * d_A, int * d_B, int * d_C)
{
    // maps local index of each thread to global index 
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   
   // computes product
   d_C[i] = d_A[i] * d_B[i];
}

